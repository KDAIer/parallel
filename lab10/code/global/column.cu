// col_matmul.cu
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void matMulCol(const float *A, const float *B, float *C,
                          int m, int n, int k) {
    // 这里我们把 threadIdx.x 用于行，threadIdx.y 用于列
    int row = blockIdx.y * blockDim.x + threadIdx.x;  
    int col = blockIdx.x * blockDim.y + threadIdx.y;  
    if (row < m && col < k) {
        float sum = 0.0f;
        for (int j = 0; j < n; ++j) {
            sum += A[row * n + j] * B[j * k + col];
        }
        C[row * k + col] = sum;
    }
}

int main(int argc, char **argv) {
    if (argc != 6) {
        printf("Usage: %s m n k block_x block_y\n", argv[0]);
        return -1;
    }
    int m = atoi(argv[1]), n = atoi(argv[2]), k = atoi(argv[3]);
    int bx = atoi(argv[4]), by = atoi(argv[5]);

    size_t sizeA = m * n * sizeof(float);
    size_t sizeB = n * k * sizeof(float);
    size_t sizeC = m * k * sizeof(float);

    float *h_A = (float*)malloc(sizeA);
    float *h_B = (float*)malloc(sizeB);
    float *h_C = (float*)malloc(sizeC);
    for (int i = 0; i < m*n; ++i) h_A[i] = rand() / (float)RAND_MAX;
    for (int i = 0; i < n*k; ++i) h_B[i] = rand() / (float)RAND_MAX;

    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, sizeA);
    hipMalloc(&d_B, sizeB);
    hipMalloc(&d_C, sizeC);
    hipMemcpy(d_A, h_A, sizeA, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, sizeB, hipMemcpyHostToDevice);

    dim3 block(bx, by);
    dim3 grid((k + bx - 1) / bx, (m + by - 1) / by);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    matMulCol<<<grid, block>>>(d_A, d_B, d_C, m, n, k);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float ms;
    hipEventElapsedTime(&ms, start, stop);
    printf("Column: %.3f ms\n", ms);

    hipFree(d_A); hipFree(d_B); hipFree(d_C);
    free(h_A); free(h_B); free(h_C);
    return 0;
}
